#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace std;

#define datafloat float

#define BDIM 256

__global__ void partialSum(const int N,
			   datafloat *u,
			   datafloat *blocksum){

  __shared__ datafloat s_blocksum[BDIM];

  int t = threadIdx.x;  
  int b = blockIdx.x;
  const int n = b*blockDim.x + t;
  
  s_blocksum[t] = 0;
  
  // prefetch one entry per thread to shared memory
  if(n < N){
    s_blocksum[t] = u[n];
  }

  // initially tag all threads as alive
  int alive = blockDim.x;

  while(alive>1){

    __syncthreads();  // barrier (make sure s_red is ready)
    
    alive /= 2;
    if(t < alive)
      s_blocksum[t] += s_blocksum[t+alive];
  }
  
  // value in s_blocksum[0] is sum of block of values
  if(t==0) 
    blocksum[b] = s_blocksum[0];
}
  

// same partial sum reduction, but with unrolled while loop
__global__ void unrolledPartialSum(const int N,
				   datafloat *u,
				   datafloat *blocksum){

  __shared__ datafloat s_blocksum[BDIM];

  int t = threadIdx.x;  
  int b = blockIdx.x;
  const int n = b*blockDim.x + t;

  s_blocksum[t] = 0;
  
  if(n<N){
    s_blocksum[t] = u[n];
  }

  __syncthreads();  // barrier (make sure s_blocksum is ready)

  // manually unrolled blocksumuction (assumes BDIM=256)
  if(BDIM>128) {
    if(t<128)
      s_blocksum[t] += s_blocksum[t+128];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>64){
    if(t<64)
      s_blocksum[t] += s_blocksum[t+64];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>32){
    if(t<32)
      s_blocksum[t] += s_blocksum[t+32];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>16){
    if(t<16)
      s_blocksum[t] += s_blocksum[t+16];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>8){
    if(t<8)
      s_blocksum[t] += s_blocksum[t+8];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>4){
    if(t<4)
      s_blocksum[t] += s_blocksum[t+4];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>2){
    if(t<2)
      s_blocksum[t] += s_blocksum[t+2];

    __syncthreads();  // barrier (make sure s_blocksum is ready)
  }

  if(BDIM>1){
    if(t<1)
      s_blocksum[t] += s_blocksum[t+1];
  }

  // store result of this block blocksumuction
  if(t==0)
    blocksum[b] = s_blocksum[t];
}

void sum(int N, datafloat *h_u){

  // Device Arrays
  datafloat *c_u, *c_partialsum;

  // Host array for partial sum
  datafloat *h_partialsum;

  // number of thread-blocks to partial sum u
  int GDIM = (N+BDIM-1)/BDIM;

  // allocate host array
  h_partialsum = (datafloat*) calloc(GDIM, sizeof(datafloat));

  // allocate device arrays
  hipMalloc((void**) &c_u  , N*sizeof(datafloat));
  hipMalloc((void**) &c_partialsum , GDIM*sizeof(datafloat));

  // copy from h_u to c_u (HOST to DEVICE)
  hipMemcpy(c_u ,  h_u ,  N*sizeof(datafloat), hipMemcpyHostToDevice);
  
  // Create CUDA events
  hipEvent_t startEvent, endEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&endEvent);

  hipEventRecord(startEvent, 0);

  // perform reduction 10 times
  int Ntests = 10, test;
  datafloat psum = 0;

  for(test=0;test<Ntests;++test){
    // perform tree wise block reduction on DEVICE
    unrolledPartialSum <<< GDIM, BDIM >>> (N, c_u, c_partialsum);

    // copy array of partially summed values to HOST
    hipMemcpy(h_partialsum, c_partialsum, GDIM*sizeof(datafloat), hipMemcpyDeviceToHost);

    // Finish reduce on host
    psum = 0;
    for(int n=0;n<GDIM;++n){
      psum += h_partialsum[n];
    }
  }

  // do timing
  hipEventRecord(endEvent, 0);
  hipEventSynchronize(endEvent);

  // Get time taken
  float timeTaken;
  hipEventElapsedTime(&timeTaken, startEvent, endEvent);

  const datafloat avgTimePerTest = timeTaken/(1000.f*(datafloat) Ntests);
  printf("average time per test = %g\n", avgTimePerTest);

  // output summation result
  printf("sum total = %g\n", psum);
  
  // free device arrays
  hipFree(c_u);
  hipFree(c_partialsum);

  // free HOST array
  free(h_partialsum);

}

int main(int argc, char** argv){

  // parse command line arguements
  if(argc != 2){
    printf("Usage: ./main N \n");
    return 0;
  }

  // Number of internal domain nodes in each direction
  const int N     = atoi(argv[1]);

  // Host Arrays
  datafloat *h_u   = (datafloat*) calloc(N, sizeof(datafloat));
  
  // initialize host array
  for(int n = 0;n < N; ++n){
    h_u[n] = 1;
  }

  // Solve discrete Laplacian
  sum(N, h_u);

  // Free the host array
  free(h_u);
}
